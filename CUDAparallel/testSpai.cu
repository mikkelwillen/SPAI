#include <stdio.h>
#include <stdlib.h>
#include "csc.cu.h"
#include "constants.cu.h"
#include "parallelSpai.cu.h"
#include "parallelTest.cu.h"
#include "qrBatched.cu.h"
#include "invBatched.cu.h"
#include "permutation.cu.h"
#include "updateQR.cu.h"


int runIdentityTest(CSC* cscA, int m, int n, float sparsity, float tolerance, int maxIterations, int s, int batchsize) {
    float* identity = (float*) malloc (sizeof(float) * n * n);

    struct CSC* res = parallelSpai(cscA, tolerance, maxIterations, s, batchsize);
    printf("After parallelSpai\n");
    int* I = (int*) malloc(sizeof(int) * m);
    int* J = (int*) malloc(sizeof(int) * n);
    for (int i = 0; i < m; i++) {
        I[i] = i;
    }
    for (int i = 0; i < n; i++) {
        J[i] = i;
    }

    float* A = CSCToDense(cscA, I, J, m, n);
    float* inv = CSCToDense(res, I, J, m, n);
    
    // identity = A * inv
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n;j++) {
            identity[i * n + j] = 0.0;
            for (int k = 0; k < n; k++) {
                identity[i * n + j] += A[i * n + k] * inv[k * n + j];
            }
        }
    }

    // print A
    printf("A:\n");
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n;j++) {
            printf("%f ", A[i * n + j]);
        }
        printf("\n");
    }

    // print inv
    printf("inv:\n");
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n;j++) {
            printf("%f ", inv[i * n + j]);
        }
        printf("\n");
    }

    // print identity
    printf("identity:\n");
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n;j++) {
            printf("%f ", identity[i * n + j]);
        }
        printf("\n");
    }

    // calculate error
    float error = 0.0;
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n;j++) {
            error += (identity[i * n + j] - (i == j ? 1.0 : 0.0)) * (identity[i * n + j] - (i == j ? 1.0 : 0.0));
        }
    }

    printf("Error: %f%\n", error);
}

int main(int argc, char** argv) {
    if (argc == 1) {
        initHwd();
        int m = 4;
        int n = 16;
        float sparsity = 1.0;
        float tolerance = 0.01;
        int maxIterations = n - 1;
        int s = 1;
        int batchsize = 4;
    
    
    
        float* A = (float*) malloc(sizeof(float) * m * n);
        float* B = (float*) malloc(sizeof(float) * m * n);
        float* C = (float*) malloc(sizeof(float) * n * n);
        float* m4 = (float*) malloc(sizeof(float) * 4 * 4);
    
        for (int i = 0; i < m; i++) {
            for (int j = 0; j < n; j++) {
                A[i * n + j] = (float) i * n + j;
            }
        }
    
        C[0] = 20.0; C[1] = 0.0;   C[2] = 0.0; 
        C[3] = 0.0;  C[4] = 30.0;  C[5] = 10.0; 
        C[6] = 0.0;  C[7] = 0.0;   C[8] = 10.0;
    
        B[0] = 20.0; B[1] = 0.0;   B[2] = 0.0; 
        B[3] = 0.0;  B[4] = 30.0;  B[5] = 10.0; 
        B[6] = 0.0;  B[7] = 0.0;   B[8] = 0.0; 
        B[9] = 0.0;  B[10] = 40.0; B[11] = 0.0;
        
        m4[0] = 10.0; m4[1] = 10.0; m4[2] = 1.2; m4[3] = 14.0;
        m4[4] = 0.0; m4[5] = 10.0; m4[6] = 2.0; m4[7] = 0.0;
        m4[8] = 13.0; m4[9] = 0.0; m4[10] = 5.3; m4[11] = 1.0;
        m4[12] = 0.0; m4[13] = 5.0; m4[14] = 0.0; m4[15] = 0.0;

        struct CSC* cscA = createCSC(A, m, n);
        struct CSC* cscB = createCSC(B, m, n);
        struct CSC* cscC = createRandomCSC(n, n, sparsity);
        struct CSC* cscD = createCSC(C, n, n);
        struct CSC* cscM4 = createCSC(m4, 4, 4);

        runIdentityTest(cscC, n, n, sparsity, tolerance, maxIterations, s, batchsize);
    } else if (argc == 8) {
        // read args
        printf("hallo?\n");
        int sizeOfMatrix = atoi(argv[1]);
        int numberOfTests = atoi(argv[2]);
        float sparsity = atof(argv[3]);
        float tolerance = atof(argv[4]);
        int maxIterations = atoi(argv[5]);
        int s = atoi(argv[6]);
        int batchsize = atoi(argv[7]);

        printf("sizeOfMatrix: %d\n", sizeOfMatrix);
        printf("numberOfTests: %d\n", numberOfTests);
        printf("sparsity: %f\n", sparsity);
        printf("tolerance: %f\n", tolerance);
        printf("maxIterations: %d\n", maxIterations);
        printf("s: %d\n", s);

        for (int i = 0; i < numberOfTests; i++) {
            CSC* csc = createRandomCSC(sizeOfMatrix, sizeOfMatrix, sparsity);

            parallelTest(csc, tolerance, maxIterations, s, batchsize);
        }
    }

    return 0;
}